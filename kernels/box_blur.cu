#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <c10/cuda/CUDAException.h>
#include <ATen/cuda/HIPContext.h>

#include "utils.cuh"

__global__ void blurKernel(unsigned char *in, unsigned char *out, int w, int h, int channels, int BLUR_SIZE) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < w && row < h) {

        for (int c = 0; c < channels; ++c) {
            int pixVal = 0;
            int pixels = 0;

            // Average of surrounding blur_size x blur_size box
            for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
                for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
                    int curRow = row + blurRow;
                    int curCol = col + blurCol;

                    if (curRow >= 0 && curRow < h && curCol >= 0 && curCol < w) {

                        pixVal += in[(curRow * w + curCol) * channels + c];
                        ++pixels;
                    }
                }
            }

            out[(row * w + col) * channels + c] = (unsigned char)(pixVal / pixels);
        }
    }
}


torch::Tensor gaussian_blur(torch::Tensor img, int blurSize) {
    assert(img.device().type() == torch::kCUDA);
    assert(img.dtype() == torch::kByte);

    const auto height = img.size(0);
    const auto width = img.size(1);
    const auto channels = img.size(2);

    dim3 dimBlock = getOptimalBlockDim(width, height);
    dim3 dimGrid(cdiv(width, dimBlock.x), cdiv(height, dimBlock.y));

    auto result = torch::empty({height, width, channels}, 
                              torch::TensorOptions().dtype(torch::kByte).device(img.device()));

    blurKernel<<<dimGrid, dimBlock, 0, at::cuda::getCurrentCUDAStream()>>>(
        img.data_ptr<unsigned char>(), 
        result.data_ptr<unsigned char>(), 
        width, height, channels, blurSize);

    C10_CUDA_KERNEL_LAUNCH_CHECK();

    return result;
}